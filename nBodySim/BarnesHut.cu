﻿#include "BarnesHut.cuh"
#include "implementKernel.cuh"

void BarnesHutGPU::InitRoot(int n,float halfSide)
{
	
	threadsPerBlock = 256;
	if (threadsPerBlock > n)
	{
		threadsPerBlock = n;
	}
	//numberOfBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,0);

	numNodes = n * 2;

	numberOfBlocks = deviceProp.multiProcessorCount;

	if (numNodes < 1024 * numberOfBlocks) numNodes = 1024 * numberOfBlocks;
	/*while ((numNodes & (WARPSIZE - 1)) != 0) nnodes++;
	nnodes--;*/
	int bytes = sizeof(NodeGPU);

	hipMallocManaged(&root, bytes);


	//hipMalloc((void**)&children, 8 * (numNodes + 1) * sizeof(int));
	bool worked = true;
	if (hipSuccess != hipMalloc((void**)&children, sizeof(int) * (numNodes + 1) * 8))
	{
		worked = false;
	}
	hipMalloc((void**)&counter, sizeof(int) * (numNodes + 1) * 8);
	hipMalloc((void**)&masses, sizeof(float) * (numNodes + 1) * 8);

	hipMalloc((void**)&cmx, sizeof(float) * (numNodes + 1) * 8);
	hipMalloc((void**)&cmy, sizeof(float) * (numNodes + 1) * 8);
	hipMalloc((void**)&cmz, sizeof(float) * (numNodes + 1) * 8);


	root->position = Vector3(halfSide, halfSide, halfSide);
	root->sideLegnth = halfSide * 2.0f;

}


void BarnesHutGPU::ConstructTree(int n, Particle* pArray)
{
	Level startLevel;
	startLevel.maxIndex = 0;
	startLevel.minIndex = 0;
	startLevel.treeLevel = 0;


	
	rootKernel << <1, 1 >> > (children, masses, numNodes);
	hipDeviceSynchronize();
	clearKernel << <threadsPerBlock, numberOfBlocks >> > (children, masses, numNodes,n);
	hipDeviceSynchronize();
	
	buildTreeInsertion << <32, 2 >> > (root, n, pArray, children,numNodes);
	
	hipDeviceSynchronize();

	//CalculateForces << <threadsPerBlock, numberOfBlocks >> > (children,counter,masses,cmx,cmy,cmz,numNodes, n, pArray,0.5f,0.5f, root->sideLegnth);
	//hipDeviceSynchronize();
	//IntegrateBH << <threadsPerBlock, numberOfBlocks >> > (n, pArray, 0.5f);
	hipDeviceSynchronize();
	int f = 0;
	f++;

	//hipFree(currentNode);
}